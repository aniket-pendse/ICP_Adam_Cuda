#include "hip/hip_runtime.h"
// HOST Code to compute ICP for localization 



#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include<vector>
#include<time.h>
#include<sys/time.h>
#include<ctime>
#include "dlib/optimization/optimization.h"
#include "dlib/optimization/find_optimal_parameters_abstract.h"
#include "dlib/optimization/optimization_bobyqa.h"
#include "dlib/optimization/find_optimal_parameters.h" 
#include "octree_code/octree.h"
#include "globals.h"
//#include "icp_gold.cpp"

extern "C"

using namespace std;

typedef dlib::matrix<double,0,1> column_vector;

__constant__ double R_constant[9];
__constant__ double t_constant[3];

#include "icp_kernel.cu"

// Function declarations
Matrix AllocateDeviceMatrix(const Matrix M);
Vector AllocateDeviceVector(const Vector V);



// Creating variables to store the measurement and model data
point_cloud_data measurement_data;
point_cloud_data model_data;

//CPU functions
dlib::matrix<double> PerformRotation(dlib::matrix<double> R,dlib::matrix<double> t, dlib::matrix<double> point);

void PerformTransformationToAllPoints(dlib::matrix<double> R, dlib::matrix<double> t, point_cloud_data * data, point_cloud_data * transformed_data, int skips);

void cal_closest_points_cpu(const column_vector &rt);

double findTotalErrorInCloud_cpu(const column_vector &rt);

dlib::matrix<double> compute_gold();
	

///// For initial testing purposes carrying out rotation and translation operation on cuda//////////////////

void cal_closest_points(Matrix rt);
double findTotalErrorInCloudOnDevice(const column_vector &rt_vec);


// Function to carry out Rotation of given point on the device 

void PerformRotationOnDevice(const Matrix R_h, const Matrix t_h, point_cloud_data * data, point_cloud_data * transformed_data)
{
	int size_data = data->size;

//***********Allocate Memory on device********************

	double * data_x_d;
	hipMalloc((void**)&data_x_d, size_data*sizeof(double));
	double * data_y_d;
	hipMalloc((void**)&data_y_d, size_data*sizeof(double));
	double * data_z_d;
	hipMalloc((void**)&data_z_d, size_data*sizeof(double));

	double * transformed_data_x_d;
	hipMalloc((void**)&transformed_data_x_d, size_data*sizeof(double));
	double * transformed_data_y_d;
	hipMalloc((void**)&transformed_data_y_d, size_data*sizeof(double));
	double * transformed_data_z_d;
	hipMalloc((void**)&transformed_data_z_d, size_data*sizeof(double));

	//Allocate temporary memory for x,y,x
	double * temp_x = (double*)malloc(size_data*sizeof(double));
	double * temp_y = (double*)malloc(size_data*sizeof(double));
	double * temp_z = (double*)malloc(size_data*sizeof(double));

//---------------------------------------------------------

//**************Copy data to Device and constant memory******
		
	hipMemcpy(data_x_d, data->x_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(data_y_d, data->y_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(data_z_d, data->z_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	
	hipMemcpyToSymbol(HIP_SYMBOL(R_constant),R_h.elements,3 * 3*sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(t_constant), t_h.elements,3*sizeof(double));
	
//----------------------------------------------------------- 
	 
//******Setup the execution configuration*********************

	dim3 block, grid;
	block.x = TILE_WIDTH;
	block.y = 1;
	block.z = 1;
	
	if(size_data%block.x == 0)
		grid.x = size_data/block.x;
	else
		grid.x = size_data/block.x + 1;
	grid.y = 1;
	grid.z = 1;

//--------------------------------------------------------------

    // Launch the device computation threads!

	PerformRotationKernel<<<grid,block>>>(data_x_d, data_y_d, data_z_d, transformed_data_x_d, transformed_data_y_d, transformed_data_z_d, size_data);
		
	// Transfer Rotated Point from device to host
	
	hipMemcpy(temp_x, transformed_data_x_d, size_data*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(temp_y, transformed_data_y_d, size_data*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(temp_z, transformed_data_z_d, size_data*sizeof(double), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < size_data; i++)
	{
		transformed_data->x_coord.push_back(temp_x[i]);
		transformed_data->y_coord.push_back(temp_y[i]);
		transformed_data->z_coord.push_back(temp_z[i]);
		transformed_data->index.push_back(-1);
	}

	//cout<<"Value in transformed data "<<transformed_data->x_coord[0]<<endl;
	//cout<<"Value in Temp x value "<<temp_x[0]<<endl;
	//cout<<"Value in input data "<<data->x_coord[0]<<endl;

	transformed_data->size = data->size;
	
	
       // Free device memory for all
       hipFree(data_x_d); hipFree (data_y_d);hipFree (data_z_d);	
	
}



// The main function 

int main()
{
	
	
	


	ifstream infile1;
  	infile1.open ("icp_model.csv");
	char* pEnd;
	string x,y,z;

	
	// Reading data from the model map data csv file 

	
	 while(!infile1.eof()){
		getline(infile1,x, ',');
		getline(infile1,y, ',');
		getline(infile1,z);
		//getline(infile,index);
		model_data.x_coord.push_back(strtod(x.c_str(),&pEnd));
		model_data.y_coord.push_back(strtod(y.c_str(),&pEnd));
		model_data.z_coord.push_back(strtod(z.c_str(),&pEnd));
		measurement_data.index.push_back(-1);
	}
	

	
	//Remove the last elements
	model_data.x_coord.pop_back();
	model_data.y_coord.pop_back();
	model_data.z_coord.pop_back();
	model_data.size = model_data.size - 1;
	
		
	//cout<<"Range x value "<<range_x<<endl;

	model_data.size = model_data.x_coord.size();
	//cout<<"model data value "<<model_data.size<<endl;
	
	//Rotational function test
	double theta = 0.03;
	double point_x = 0.003;
	double point_y = 0.005;
	double point_z = 0.0;
	Matrix R;
	Matrix t;
	R.width = 3;
	R.height = 3;
	t.width = 1;
	t.height = 3;
	// Allocating memory to the matrices 

	R.elements = (double*)malloc(R.width*R.height*sizeof(double));
	t.elements = (double*)malloc(t.width*t.height*sizeof(double));

	R.elements[0] = cos(theta);R.elements[1]= -sin(theta); R.elements[2]= 0;
	R.elements[3] =sin(theta);  R.elements[4]=cos(theta); R.elements[5]= 0;
	R.elements[6] = 0; R.elements[7]= 0; R.elements[8]= 1;
	
	t.elements[0] = point_x;
	t.elements[1]= point_y;
	t.elements[2] = point_z;
	
	
	// Generate mesasurement datra by rorating the model data
	PerformRotationOnDevice(R, t, &model_data, &measurement_data);

	


	//Calling closest point.
	Matrix  rt;
	rt.width =  1;
	rt.height =  4;
	column_vector rt_lower(4), rt_upper(4), rt_vec(4);
	rt.elements = (double*)malloc(rt.width*rt.height*sizeof(double));
	rt.elements[0] = 0.0;rt.elements[1] = 0.0;
	rt.elements[2] = 0.0;rt.elements[3] = 0;
	
	rt_vec = rt.elements[0], rt.elements[1], rt.elements[2], rt.elements[3];	
	rt_lower = -1.0, -1.0,-1.0,-1.0;
	rt_upper = 1.0, 1.0, 1.0, 1.0;

	double temp_error = 0;	
	double cpu_starttime , cpu_endtime;
	//temp_error = findTotalErrorInCloudOnDevice(rt_vec);
	//cpu_starttime = clock();
	//cal_closest_points(rt);
	//cpu_endtime = clock();
	//cout<<"The time taken for calculation of closest point = "<<((cpu_endtime - cpu_starttime)/CLOCKS_PER_SEC)<<endl;

	double final_error = 0;
	// time measurement variables 

	column_vector rt_gold(5);
	rt_gold = compute_gold();
	
	cout<<"Rt gold "<<rt_gold<<endl;


	cpu_starttime = clock();
	//double cpu_starttime , cpu_endtime;
	for(int i = 0; i<20; i++)
	{
		cout<<"iteration #: "<<i<<endl;
		
		cal_closest_points(rt);
		final_error = find_optimal_parameters(0.01, 0.000000001,100000, rt_vec, rt_lower, rt_upper,findTotalErrorInCloudOnDevice);
		//final_error = findTotalErrorInCloudOnDevice(rt_vec);
		
		cout<<"Rt parameters "<<rt_vec<<endl;
		cout<<"current error: "<<final_error<<endl;
		rt.elements[0] = rt_vec(0);
		rt.elements[1] = rt_vec(1);
		rt.elements[2] = rt_vec(2);
		rt.elements[3] = rt_vec(3);
		
	}
	cpu_endtime = clock();
	cout<<"The time taken for calculation = "<<((cpu_endtime - cpu_starttime)/CLOCKS_PER_SEC)<<endl;
	//cout<<"Error after optimization "<<final_error<<endl;
	





	return 0;
}


//Calculating the closest point
void cal_closest_points(Matrix rt)
{
	point_cloud_data transformed_data;

	Matrix R_h, t_h;
	R_h.width = 3;
	R_h.height = 3;
	t_h.width = 1;
	t_h.height = 3;
	// Allocating memory to the matrices 

	R_h.elements = (double*)malloc(R_h.width*R_h.height*sizeof(double));
	t_h.elements = (double*)malloc(t_h.width*t_h.height*sizeof(double));

	R_h.elements[0] = cos(rt.elements[0]);R_h.elements[1]= -sin(rt.elements[0]); R_h.elements[2]= 0;
	R_h.elements[3] = sin(rt.elements[0]);  R_h.elements[4]= cos(rt.elements[0]); R_h.elements[5]= 0;
	R_h.elements[6] = 0; R_h.elements[7]= 0; R_h.elements[8]= 1;
	
	t_h.elements[0] = rt.elements[1];
	t_h.elements[1] = rt.elements[2];
	t_h.elements[2] = rt.elements[3];


	PerformRotationOnDevice(R_h, t_h, &measurement_data, &transformed_data);

	//cout<<"x of measurement data "<<transformed_data.x_coord[0]<<endl;
	//cout<<"y of measurement data "<<transformed_data.y_coord[0]<<endl;
	//cout<<"z of measurement data "<<transformed_data.z_coord[0]<<endl;		

	//cout<<"x of transformed data "<<transformed_data.x_coord[0]<<endl;
	//cout<<"y of transformed data "<<transformed_data.y_coord[0]<<endl;
	//cout<<"z of transformed data "<<transformed_data.z_coord[0]<<endl;

	//Calculate the closest point
	double * x_coord_model_d;
	hipMalloc((void**)&x_coord_model_d, model_data.size*sizeof(double));
	hipMemcpy(x_coord_model_d, model_data.x_coord.data(), model_data.size*sizeof(double), hipMemcpyHostToDevice);
	
	double * y_coord_model_d;
	hipMalloc((void**)&y_coord_model_d, model_data.size*sizeof(double));
	hipMemcpy(y_coord_model_d, model_data.y_coord.data(), model_data.size*sizeof(double), hipMemcpyHostToDevice);
	
	double * z_coord_model_d;
	hipMalloc((void**)&z_coord_model_d, model_data.size*sizeof(double));
	hipMemcpy(z_coord_model_d, model_data.z_coord.data(), model_data.size*sizeof(double), hipMemcpyHostToDevice);
	
	double * distance_d;
	hipMalloc((void**)&distance_d, model_data.size*sizeof(double));

	int * index_d;	
	hipMalloc((void**)&index_d, model_data.size*sizeof(int));

	
	
	for(int i = 0; i < transformed_data.size; i++)
	{	
		dim3 block, grid;
		block.x = TILE_WIDTH;
		block.y = 1;
		block.z = 1;
		if(transformed_data.size%block.x == 0)
			grid.x = transformed_data.size/block.x;
		else
			grid.x = transformed_data.size/block.x + 1;
		grid.y = 1;
		grid.z = 1;
		//cout<<"Check grid "<<grid.x<<endl;
		int size_data = model_data.size;
		double point_x = transformed_data.x_coord[i];
		double point_y = transformed_data.y_coord[i];
		double point_z = transformed_data.z_coord[i];
		
		CalculateDistanceIndexEachPoint<<<grid, block>>>(point_x, point_y, point_z, x_coord_model_d, y_coord_model_d, z_coord_model_d, index_d, distance_d, size_data);
	
		
		while(grid.x > 1)
		{	
			hipDeviceSynchronize();
			//cout<<"Check grid 2 "<<grid.x<<endl;
			
			if(size_data%block.x == 0)
				grid.x = size_data/block.x;
			else
				grid.x = size_data/block.x + 1;
			
			CalculateBestIndex<<<grid,block>>>(distance_d, index_d, size_data);
			size_data = grid.x;
		}		
		hipMemcpy(measurement_data.index.data() + i, index_d, sizeof(int), hipMemcpyDeviceToHost);
	}

	cout<<"Index at some point "<<measurement_data.index[10]<<endl;
	hipFree(x_coord_model_d);
	hipFree(y_coord_model_d);
	hipFree(z_coord_model_d);
	hipFree(distance_d);
	hipFree(index_d);
		



}









// Function to find the total error in cloud






double findTotalErrorInCloudOnDevice(const column_vector &rt_vec) 
{
	//iterations++;
	double icp_error = 0.0;
	point_cloud_data transformed_data;
	Matrix R, t;
        R.width = 3;R.height =3;t.height =3;t.width = 1;
	R.elements = (double*)malloc(R.width*R.height*sizeof(double));
	t.elements = (double*)malloc(t.width*t.height*sizeof(double));

	
	R.elements[0] = cos(rt_vec(0));R.elements[1] = -sin(rt_vec(0));R.elements[2] = 0; R.elements[3] = sin(rt_vec(0));R.elements[4] = cos(rt_vec(0));R.elements[5] = 0;
	R.elements[6] = 0; R.elements[7] = 0; R.elements[8] = 1;
	t.elements[0] = rt_vec(1);
	t.elements[1] =  rt_vec(2);
	t.elements[2] =  rt_vec(3);
	//cout<<"Check measurement data element "<<measurement_data.x_coord.at(0)<<endl;
	PerformRotationOnDevice(R, t, &measurement_data, &transformed_data);


	// Creating device variables 
	double * data_x_d;
	double * data_y_d;
	double * data_z_d;
	int * index_d;
	double * transformed_data_x_d;
	double * transformed_data_y_d;
	double * transformed_data_z_d;
	double * distance_d;	
	int size_data = transformed_data.size;
		


	hipMalloc((void**)&data_x_d, size_data*sizeof(double));
	hipMalloc((void**)&data_y_d, size_data*sizeof(double));
	hipMalloc((void**)&data_z_d, size_data*sizeof(double));
	hipMalloc((void**)&transformed_data_x_d, size_data*sizeof(double));
	hipMalloc((void**)&transformed_data_y_d, size_data*sizeof(double));
	hipMalloc((void**)&transformed_data_z_d, size_data*sizeof(double));
	hipMalloc((void**)&distance_d, size_data*sizeof(double));
	hipMalloc((void**)&index_d, size_data*sizeof(int));


	hipMemcpy(data_x_d, model_data.x_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(data_y_d, model_data.y_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(data_z_d, model_data.z_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(transformed_data_x_d, transformed_data.x_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(transformed_data_y_d, transformed_data.y_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(transformed_data_z_d, transformed_data.z_coord.data(), size_data*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(index_d, measurement_data.index.data(), size_data*sizeof(int), hipMemcpyHostToDevice);


	dim3 block, grid;
	block.x = TILE_WIDTH;
	block.y = 1;
	block.z = 1;
	if(size_data%block.x == 0)
		grid.x = size_data/block.x;
	else
		grid.x = size_data/block.x + 1;
	grid.y = 1;
	grid.z = 1;

	CalculateDistanceAllPoints<<<grid, block>>>(data_x_d, data_y_d, data_z_d, transformed_data_x_d, transformed_data_y_d, transformed_data_z_d, index_d, distance_d, size_data);
	while(grid.x > 1)
	{	
		hipDeviceSynchronize();
		if(size_data%block.x == 0)
			grid.x = size_data/block.x;
		else
			grid.x = size_data/block.x + 1;

		CalculateTotalError<<<grid,block>>>(distance_d, size_data);
		size_data = grid.x;
	}
	
	hipMemcpy(&icp_error, distance_d, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(data_x_d);
	hipFree(data_y_d);
	hipFree(data_z_d);
	hipFree(transformed_data_x_d);
	hipFree(transformed_data_y_d);
	hipFree(transformed_data_z_d);
	hipFree(index_d);
	hipFree(distance_d);
	
	return icp_error;
}

// Function to allocate matrix memory on the device
 
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    
    return Mdevice;
}
/*
Vector AllocateDeviceVector(std::vector<int> V)
{
    std::vector<int> Vdevice = V;
    int size = V.size() * sizeof(int);
    hipMalloc((void**)&Vdevice, size);
    return Vdevice;
}

*/












	
